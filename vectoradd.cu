#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void VecAdd(float* A, float* B, float* C) {
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

int main() {
    // Vector size
    int N = 1024;
    size_t size = N * sizeof(float);

    // Host vectors
    float *h_A, *h_B, *h_C;

    // Device vectors
    float *d_A, *d_B, *d_C;

    // Allocate memory for host vectors
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);

    // Initialize host vectors
    for (int i = 0; i < N; i++) {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    // Allocate memory for device vectors
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy host vectors to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Kernel invocation with N threads
    VecAdd<<<1, N>>>(d_A, d_B, d_C);

    // Check for error in kernel launch
    hipError_t err = hipGetLastError();
    if (err  != hipSuccess) {
        printf("Kernel launch error: %s\n", hipGetErrorString(err));
        return -1;
    }

    // Wait for kernel to finish
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify the result
    bool success = true;
    for (int i = 0; i < N; ++i) {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
            printf("Verification failed at index %d!\n", i);
            success = false;
            break;
        }
    }

    if (success) {
        printf("Vector addition successful!\n");
    }

    // Print first few results as sample
    printf("Sample results:\n");
    for (int i = 0; i < 5; i++) {
        printf("A[%d] = %.2f, B[%d] = %.2f, C[%d] = %.2f\n",
            i, h_A[i], i, h_B[i], i, h_C[i]);
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}